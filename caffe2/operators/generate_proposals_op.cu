#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "caffe2/core/context.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/generate_proposals_op.h"
#include "caffe2/operators/generate_proposals_op_util_boxes.h" // BBOX_XFORM_CLIP_DEFAULT
#include "caffe2/operators/generate_proposals_op_util_nms.h"
#include "caffe2/operators/generate_proposals_op_util_nms_gpu.h"

namespace caffe2 {
namespace {
__global__ void GeneratePreNMSUprightBoxesKernel(
    const int* d_sorted_scores_keys,
    const int nboxes_to_generate,
    const float* d_bbox_deltas,
    const float4* d_anchors,
    const int H,
    const int W,
    const int K, // K = H*W
    const int A,
    const int KA, // KA = K*A
    const float feat_stride,
    const float min_size,
    const float* d_img_info_vec,
    const int num_images,
    const float bbox_xform_clip,
    const bool correct_transform,
    float4* d_out_boxes,
    const int prenms_nboxes, // leading dimension of out_boxes
    float* d_inout_scores,
    char* d_boxes_keep_flags) {
  CUDA_2D_KERNEL_LOOP(ibox, nboxes_to_generate, image_index, num_images) {
    // box_conv_index : # of the same box, but indexed in
    // the scores from the conv layer, of shape (A,H,W)
    // the num_images dimension was already removed
    // box_conv_index = a*K + h*W + w
    const int box_conv_index = d_sorted_scores_keys[image_index * KA + ibox];

    // We want to decompose box_conv_index in (a,h,w)
    // such as box_conv_index = a*K + h*W + w
    // (avoiding modulos in the process)
    int remaining = box_conv_index;
    const int dA = K; // stride of A
    const int a = remaining / dA;
    remaining -= a * dA;
    const int dH = W; // stride of H
    const int h = remaining / dH;
    remaining -= h * dH;
    const int w = remaining; // dW = 1

    // Loading the anchor a
    // float is a struct with float x,y,z,w
    const float4 anchor = d_anchors[a];
    // x1,y1,x2,y2 :coordinates of anchor a, shifted for position (h,w)
    const float shift_w = feat_stride * w;
    float x1 = shift_w + anchor.x;
    float x2 = shift_w + anchor.z;
    const float shift_h = feat_stride * h;
    float y1 = shift_h + anchor.y;
    float y2 = shift_h + anchor.w;

    // TODO use fast math when possible

    // Deltas for that box
    // Deltas of shape (num_images,4*A,K)
    // We're going to compute 4 scattered reads
    // better than the alternative, ie transposing the complete deltas
    // array first
    int deltas_idx = image_index * (KA * 4) + a * 4 * K + h * W + w;
    const float dx = d_bbox_deltas[deltas_idx];
    // Stride of K between each dimension
    deltas_idx += K;
    const float dy = d_bbox_deltas[deltas_idx];
    deltas_idx += K;
    float dw = d_bbox_deltas[deltas_idx];
    deltas_idx += K;
    float dh = d_bbox_deltas[deltas_idx];

    // Upper bound on dw,dh
    dw = fmin(dw, bbox_xform_clip);
    dh = fmin(dh, bbox_xform_clip);

    // Applying the deltas
    float width = x2 - x1 + 1.0f;
    const float ctr_x = x1 + 0.5f * width;
    const float pred_ctr_x = ctr_x + width * dx; // TODO fuse madd
    const float pred_w = width * expf(dw);
    x1 = pred_ctr_x - 0.5f * pred_w;
    x2 = pred_ctr_x + 0.5f * pred_w;

    float height = y2 - y1 + 1.0f;
    const float ctr_y = y1 + 0.5f * height;
    const float pred_ctr_y = ctr_y + height * dy;
    const float pred_h = height * expf(dh);
    y1 = pred_ctr_y - 0.5f * pred_h;
    y2 = pred_ctr_y + 0.5f * pred_h;

    if (correct_transform) {
      x2 -= 1.0f;
      y2 -= 1.0f;
    }

    // Clipping box to image
    const float img_height = d_img_info_vec[3 * image_index + 0];
    const float img_width = d_img_info_vec[3 * image_index + 1];
    const float min_size_scaled =
        min_size * d_img_info_vec[3 * image_index + 2];
    x1 = fmax(fmin(x1, img_width - 1.0f), 0.0f);
    y1 = fmax(fmin(y1, img_height - 1.0f), 0.0f);
    x2 = fmax(fmin(x2, img_width - 1.0f), 0.0f);
    y2 = fmax(fmin(y2, img_height - 1.0f), 0.0f);

    // Filter boxes
    // Removing boxes with one dim < min_size
    // (center of box is in image, because of previous step)
    width = x2 - x1 + 1.0f; // may have changed
    height = y2 - y1 + 1.0f;
    bool keep_box = fmin(width, height) >= min_size_scaled;

    // We are not deleting the box right now even if !keep_box
    // we want to keep the relative order of the elements stable
    // we'll do it in such a way later
    // d_boxes_keep_flags size: (num_images,prenms_nboxes)
    // d_out_boxes size: (num_images,prenms_nboxes)
    const int out_index = image_index * prenms_nboxes + ibox;
    d_boxes_keep_flags[out_index] = keep_box;
    d_out_boxes[out_index] = {x1, y1, x2, y2};

    // d_inout_scores size: (num_images,KA)
    if (!keep_box)
      d_inout_scores[image_index * KA + ibox] = FLT_MIN; // for NMS
  }
}

__global__ void WriteOutput(
    const float4* d_image_boxes,
    const float* d_image_scores,
    const int* d_image_boxes_keep_list,
    const int nboxes,
    const int image_index,
    float* d_image_out_rois,
    float* d_image_out_rois_probs) {
  CUDA_1D_KERNEL_LOOP(i, nboxes) {
    const int ibox = d_image_boxes_keep_list[i];
    const float4 box = d_image_boxes[ibox];
    const float score = d_image_scores[ibox];
    // Scattered memory accesses
    // postnms_nboxes is small anyway
    d_image_out_rois_probs[i] = score;
    const int base_idx = 5 * i;
    d_image_out_rois[base_idx + 0] = image_index;
    d_image_out_rois[base_idx + 1] = box.x;
    d_image_out_rois[base_idx + 2] = box.y;
    d_image_out_rois[base_idx + 3] = box.z;
    d_image_out_rois[base_idx + 4] = box.w;
  }
}

__global__ void InitializeDataKernel(
    const int num_images,
    const int KA,
    int* d_image_offsets,
    int* d_boxes_keys_iota) {
  CUDA_2D_KERNEL_LOOP(box_idx, KA, img_idx, num_images) {
    d_boxes_keys_iota[img_idx * KA + box_idx] = box_idx;

    // One 1D line sets the 1D data
    if (box_idx == 0) {
      d_image_offsets[img_idx] = KA * img_idx;
      // One thread sets the last+1 offset
      if (img_idx == 0)
        d_image_offsets[num_images] = KA * num_images;
    }
  }
}

} // namespace

template <>
bool GenerateProposalsOp<HIPContext>::RunOnDevice() {
  const auto& scores = Input(0);
  const auto& bbox_deltas = Input(1);
  const auto& im_info_tensor = Input(2);
  const auto& anchors = Input(3);
  auto* out_rois = Output(0);
  auto* out_rois_probs = Output(1);

  CAFFE_ENFORCE_EQ(scores.ndim(), 4, scores.ndim());
  CAFFE_ENFORCE(scores.template IsType<float>(), scores.meta().name());

  const auto num_images = scores.dim(0);
  const auto A = scores.dim(1);
  const auto H = scores.dim(2);
  const auto W = scores.dim(3);
  const auto box_dim_conv = anchors.dim(1);

  CAFFE_ENFORCE(box_dim_conv == 4); // only upright boxes in GPU version for now

  constexpr int box_dim = 4;
  const int K = H * W;
  const int conv_layer_nboxes = K * A;
  // Getting data members ready

  // We'll sort the scores
  // we want to remember their original indexes,
  // ie their indexes in the tensor of shape (num_images,A,K)
  // from the conv layer
  // each row of d_conv_layer_indexes is at first initialized to 1..A*K
  dev_conv_layer_indexes_.Resize(num_images, conv_layer_nboxes);
  int* d_conv_layer_indexes =
      dev_conv_layer_indexes_.template mutable_data<int>();

  // d_image_offset[i] = i*K*A for i from 1 to num_images+1
  // Used by the segmented sort to only sort scores within one image
  dev_image_offset_.Resize(num_images + 1);
  int* d_image_offset = dev_image_offset_.template mutable_data<int>();

  // The following calls to CUB primitives do nothing
  // (because the first arg is nullptr)
  // except setting cub_*_temp_storage_bytes
  size_t cub_sort_temp_storage_bytes = 0;
  float* flt_ptr = nullptr;
  int* int_ptr = nullptr;
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      nullptr,
      cub_sort_temp_storage_bytes,
      flt_ptr,
      flt_ptr,
      int_ptr,
      int_ptr,
      num_images * conv_layer_nboxes,
      num_images,
      int_ptr,
      int_ptr,
      0,
      8 * sizeof(float), // sort all bits
      context_.hip_stream());

  // Allocate temporary storage for CUB
  dev_cub_sort_buffer_.Resize(cub_sort_temp_storage_bytes);
  void* d_cub_sort_temp_storage =
      dev_cub_sort_buffer_.template mutable_data<char>();

  size_t cub_select_temp_storage_bytes = 0;
  char* char_ptr = nullptr;
  hipcub::DeviceSelect::Flagged(
      nullptr,
      cub_select_temp_storage_bytes,
      flt_ptr,
      char_ptr,
      flt_ptr,
      int_ptr,
      K * A,
      context_.hip_stream());

  // Allocate temporary storage for CUB
  dev_cub_select_buffer_.Resize(cub_select_temp_storage_bytes);
  void* d_cub_select_temp_storage =
      dev_cub_select_buffer_.template mutable_data<char>();

  // Initialize :
  // - each row of dev_conv_layer_indexes to 1..K*A
  // - each d_nboxes to 0
  // - d_image_offset[i] = K*A*i for i 1..num_images+1
  // 2D grid
  InitializeDataKernel<<<
      (CAFFE_GET_BLOCKS(A * K), num_images),
      CAFFE_CUDA_NUM_THREADS, // blockDim.y == 1
      0,
      context_.hip_stream()>>>(
      num_images, conv_layer_nboxes, d_image_offset, d_conv_layer_indexes);

  // Sorting input scores
  dev_sorted_conv_layer_indexes_.Resize(num_images, conv_layer_nboxes);
  dev_sorted_scores_.Resize(num_images, conv_layer_nboxes);
  const float* d_in_scores = scores.data<float>();
  int* d_sorted_conv_layer_indexes =
      dev_sorted_conv_layer_indexes_.template mutable_data<int>();
  float* d_sorted_scores = dev_sorted_scores_.template mutable_data<float>();
  ;
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      d_cub_sort_temp_storage,
      cub_sort_temp_storage_bytes,
      d_in_scores,
      d_sorted_scores,
      d_conv_layer_indexes,
      d_sorted_conv_layer_indexes,
      num_images * conv_layer_nboxes,
      num_images,
      d_image_offset,
      d_image_offset + 1,
      0,
      8 * sizeof(float), // sort all bits
      context_.hip_stream());

  // Keeping only the topN pre_nms
  const int nboxes_to_generate = std::min(conv_layer_nboxes, rpn_pre_nms_topN_);

  // Generating the boxes associated to the topN pre_nms scores
  dev_boxes_.Resize(num_images, box_dim * nboxes_to_generate);
  dev_boxes_keep_flags_.Resize(num_images, nboxes_to_generate);
  const float* d_bbox_deltas = bbox_deltas.data<float>();
  const float* d_anchors = anchors.data<float>();
  const float* d_im_info_vec = im_info_tensor.data<float>();
  float4* d_boxes =
      reinterpret_cast<float4*>(dev_boxes_.template mutable_data<float>());
  ;
  char* d_boxes_keep_flags =
      dev_boxes_keep_flags_.template mutable_data<char>();

  GeneratePreNMSUprightBoxesKernel<<<
      (CAFFE_GET_BLOCKS(nboxes_to_generate), num_images),
      CAFFE_CUDA_NUM_THREADS, // blockDim.y == 1
      0,
      context_.hip_stream()>>>(
      d_sorted_conv_layer_indexes,
      nboxes_to_generate,
      d_bbox_deltas,
      reinterpret_cast<const float4*>(d_anchors),
      H,
      W,
      K,
      A,
      K * A,
      feat_stride_,
      rpn_min_size_,
      d_im_info_vec,
      num_images,
      utils::BBOX_XFORM_CLIP_DEFAULT,
      correct_transform_coords_,
      d_boxes,
      nboxes_to_generate,
      d_sorted_scores,
      d_boxes_keep_flags);
  const int nboxes_generated = nboxes_to_generate;
  dev_image_prenms_boxes_.Resize(box_dim * nboxes_generated);
  float4* d_image_prenms_boxes = reinterpret_cast<float4*>(
      dev_image_prenms_boxes_.template mutable_data<float>());
  dev_image_prenms_scores_.Resize(nboxes_generated);
  float* d_image_prenms_scores =
      dev_image_prenms_scores_.template mutable_data<float>();
  dev_image_boxes_keep_list_.Resize(nboxes_generated);
  int* d_image_boxes_keep_list =
      dev_image_boxes_keep_list_.template mutable_data<int>();

  const int max_postnms_nboxes = std::min(nboxes_generated, rpn_post_nms_topN_);
  dev_postnms_rois_.Resize(5 * num_images * max_postnms_nboxes);
  dev_postnms_rois_probs_.Resize(num_images * max_postnms_nboxes);
  float* d_postnms_rois = dev_postnms_rois_.template mutable_data<float>();
  float* d_postnms_rois_probs =
      dev_postnms_rois_probs_.template mutable_data<float>();

  dev_prenms_nboxes_.Resize(num_images);
  host_prenms_nboxes_.Resize(num_images);
  int* d_prenms_nboxes = dev_prenms_nboxes_.template mutable_data<int>();
  int* h_prenms_nboxes = host_prenms_nboxes_.template mutable_data<int>();

  int nrois_in_output = 0;
  for (int image_index = 0; image_index < num_images; ++image_index) {
    // Sub matrices for current image
    const float4* d_image_boxes = &d_boxes[image_index * nboxes_generated];
    const float* d_image_sorted_scores = &d_sorted_scores[image_index * K * A];
    char* d_image_boxes_keep_flags =
        &d_boxes_keep_flags[image_index * nboxes_generated];

    float* d_image_postnms_rois = &d_postnms_rois[5 * nrois_in_output];
    float* d_image_postnms_rois_probs = &d_postnms_rois_probs[nrois_in_output];

    // Moving valid boxes (ie the ones with d_boxes_keep_flags[ibox] == true)
    // to the output tensors

    hipcub::DeviceSelect::Flagged(
        d_cub_select_temp_storage,
        cub_select_temp_storage_bytes,
        d_image_boxes,
        d_image_boxes_keep_flags,
        d_image_prenms_boxes,
        d_prenms_nboxes,
        nboxes_generated,
        context_.hip_stream());

    hipcub::DeviceSelect::Flagged(
        d_cub_select_temp_storage,
        cub_select_temp_storage_bytes,
        d_image_sorted_scores,
        d_image_boxes_keep_flags,
        d_image_prenms_scores,
        d_prenms_nboxes,
        nboxes_generated,
        context_.hip_stream());

    host_prenms_nboxes_.CopyFrom(dev_prenms_nboxes_);

    // We know prenms_boxes <= topN_prenms, because nboxes_generated <=
    // topN_prenms Calling NMS on the generated boxes
    const int prenms_nboxes = *h_prenms_nboxes;
    int nkeep;
    utils::nms_gpu_upright(
        reinterpret_cast<const float*>(d_image_prenms_boxes),
        prenms_nboxes,
        rpn_nms_thresh_,
        d_image_boxes_keep_list,
        &nkeep,
        dev_nms_mask_,
        host_nms_mask_,
        &context_);

    // All operations done after previous sort were keeping the relative order
    // of the elements the elements are still sorted keep topN <=> truncate the
    // array
    const int postnms_nboxes = std::min(nkeep, rpn_post_nms_topN_);

    // Moving the out boxes to the output tensors,
    // adding the image_index dimension on the fly
    WriteOutput<<<
        CAFFE_GET_BLOCKS(postnms_nboxes),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        d_image_prenms_boxes,
        d_image_prenms_scores,
        d_image_boxes_keep_list,
        postnms_nboxes,
        image_index,
        d_image_postnms_rois,
        d_image_postnms_rois_probs);

    nrois_in_output += postnms_nboxes;
  }

  // Using a buffer because we cannot call ShrinkTo
  out_rois->Resize(nrois_in_output, 5);
  out_rois_probs->Resize(nrois_in_output);
  float* d_out_rois = out_rois->template mutable_data<float>();
  float* d_out_rois_probs = out_rois_probs->template mutable_data<float>();

  CUDA_CHECK(hipMemcpyAsync(
      d_out_rois,
      d_postnms_rois,
      nrois_in_output * 5 * sizeof(float),
      hipMemcpyDeviceToDevice,
      context_.hip_stream()));
  CUDA_CHECK(hipMemcpyAsync(
      d_out_rois_probs,
      d_postnms_rois_probs,
      nrois_in_output * sizeof(float),
      hipMemcpyDeviceToDevice,
      context_.hip_stream()));

  return true;
}

REGISTER_HIP_OPERATOR(GenerateProposals, GenerateProposalsOp<HIPContext>);
} // namespace caffe2
