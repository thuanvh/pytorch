#include "hip/hip_runtime.h"
// TODO(ataei): reduce the apparent redundancy of all the code below.
#include "caffe2/operators/pool_op.h"

#include <cfloat>
#include <functional>
#include <numeric>

#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

struct AveragePool {
  explicit AveragePool(const OperatorBase& /* op */) {}
};

struct MaxPool {
  explicit MaxPool(const OperatorBase& /* op */) {}
};

template <typename T>
__global__ void AveragePool1DForwardNCHWCUDAKernel(
    const int K,
    const int X_size,
    const int Y_size,
    const int kernel,
    const int stride,
    const int pad,
    const bool count_include_pad,
    const T* X,
    T* Y) {
  const int nc = blockIdx.x / K;
  const int block = blockIdx.x % K;
  const T* X_ptr = X + nc * X_size;
  T* Y_ptr = Y + nc * Y_size;
  const int y = threadIdx.x + block * CAFFE_CUDA_NUM_THREADS;
  if (y < Y_size) {
    const int x = y * stride;
    const int l = max(x - pad, 0);
    const int r = min(x - pad + kernel, X_size);
    const T scale = T(1) / static_cast<T>(count_include_pad ? kernel : r - l);
    T sum = 0;
    for (int i = l; i < r; ++i) {
      sum += X_ptr[i];
    }
    Y_ptr[y] = sum * scale;
  }
}

template <typename T>
__global__ void AveragePool1DForwardNHWCCUDAKernel(
    const int C,
    const int X_size,
    const int Y_size,
    const int kernel,
    const int stride,
    const int pad,
    const bool count_include_pad,
    const T* X,
    T* Y) {
  const int n = blockIdx.x / Y_size;
  const int y = blockIdx.x % Y_size;
  const int x = y * stride;
  const int l = max(x - pad, 0);
  const int r = min(x - pad + kernel, X_size);
  const T scale = T(1) / static_cast<T>(count_include_pad ? kernel : r - l);
  const T* X_ptr = X + n * X_size * C;
  T* Y_ptr = Y + n * Y_size * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T sum = 0;
    for (int i = l; i < r; ++i) {
      sum += X_ptr[i * C + c];
    }
    Y_ptr[y * C + c] = sum * scale;
  }
}

template <typename T>
__global__ void AveragePool2DForwardNCHWCUDAKernel(
    const int K,
    const int X_H,
    const int X_W,
    const int Y_H,
    const int Y_W,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    const bool count_include_pad,
    const T* X,
    T* Y) {
  const int X_HxW = X_H * X_W;
  const int Y_HxW = Y_H * Y_W;
  const int nc = blockIdx.x / K;
  const int block = blockIdx.x % K;
  const T* X_ptr = X + nc * X_HxW;
  T* Y_ptr = Y + nc * Y_HxW;
  const int y = threadIdx.x + block * CAFFE_CUDA_NUM_THREADS;
  if (y < Y_HxW) {
    const int yh = y / Y_W;
    const int yw = y % Y_W;
    const int xh = yh * stride_h;
    const int xw = yw * stride_w;
    const int t = max(xh - pad_t, 0);
    const int b = min(xh - pad_t + kernel_h, X_H);
    const int l = max(xw - pad_l, 0);
    const int r = min(xw - pad_l + kernel_w, X_W);
    const T scale = T(1) /
        static_cast<T>(count_include_pad ? kernel_h * kernel_w
                                         : (b - t) * (r - l));
    T sum = 0;
    for (int i = t; i < b; ++i) {
      for (int j = l; j < r; ++j) {
        sum += X_ptr[i * X_W + j];
      }
    }
    Y_ptr[y] = sum * scale;
  }
}

template <typename T>
__global__ void AveragePool2DForwardNHWCCUDAKernel(
    const int C,
    const int X_H,
    const int X_W,
    const int Y_H,
    const int Y_W,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    const bool count_include_pad,
    const T* X,
    T* Y) {
  const int X_HxW = X_H * X_W;
  const int Y_HxW = Y_H * Y_W;
  const int n = blockIdx.x / Y_HxW;
  const int y = blockIdx.x % Y_HxW;
  const int yh = y / Y_W;
  const int yw = y % Y_W;
  const int xh = yh * stride_h;
  const int xw = yw * stride_w;
  const int t = max(xh - pad_t, 0);
  const int b = min(xh - pad_t + kernel_h, X_H);
  const int l = max(xw - pad_l, 0);
  const int r = min(xw - pad_l + kernel_w, X_W);
  const T scale = T(1) /
      static_cast<T>(count_include_pad ? kernel_h * kernel_w
                                       : (b - t) * (r - l));
  const T* X_ptr = X + n * X_HxW * C;
  T* Y_ptr = Y + n * Y_HxW * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T sum = 0;
    for (int i = t; i < b; ++i) {
      for (int j = l; j < r; ++j) {
        sum += X_ptr[(i * X_W + j) * C + c];
      }
    }
    Y_ptr[y * C + c] = sum * scale;
  }
}

template <typename T>
__global__ void AveragePool3DForwardNCHWCUDAKernel(
    const int K,
    const int X_D,
    const int X_H,
    const int X_W,
    const int Y_D,
    const int Y_H,
    const int Y_W,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_p,
    const int pad_t,
    const int pad_l,
    const bool count_include_pad,
    const T* X,
    T* Y) {
  const int X_HxW = X_D * X_H * X_W;
  const int Y_HxW = Y_D * Y_H * Y_W;
  const int nc = blockIdx.x / K;
  const int block = blockIdx.x % K;
  const T* X_ptr = X + nc * X_HxW;
  T* Y_ptr = Y + nc * Y_HxW;
  const int y = threadIdx.x + block * CAFFE_CUDA_NUM_THREADS;
  if (y < Y_HxW) {
    const int yy = y / Y_W;
    const int yw = y % Y_W;
    const int yh = yy % Y_H;
    const int yd = yy / Y_H;
    const int xd = yd * stride_d;
    const int xh = yh * stride_h;
    const int xw = yw * stride_w;
    const int p = max(xd - pad_p, 0);
    const int a = min(xd - pad_p + kernel_d, X_D);
    const int t = max(xh - pad_t, 0);
    const int b = min(xh - pad_t + kernel_h, X_H);
    const int l = max(xw - pad_l, 0);
    const int r = min(xw - pad_l + kernel_w, X_W);
    const T scale = T(1) /
        static_cast<T>(count_include_pad ? kernel_d * kernel_h * kernel_w
                                         : (a - p) * (b - t) * (r - l));
    T sum = 0;
    for (int i = p; i < a; ++i) {
      for (int j = t; j < b; ++j) {
        for (int k = l; k < r; ++k) {
          sum += X_ptr[(i * X_H + j) * X_W + k];
        }
      }
    }
    Y_ptr[y] = sum * scale;
  }
}

template <typename T>
__global__ void AveragePool3DForwardNHWCCUDAKernel(
    const int C,
    const int X_D,
    const int X_H,
    const int X_W,
    const int Y_D,
    const int Y_H,
    const int Y_W,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_p,
    const int pad_t,
    const int pad_l,
    const bool count_include_pad,
    const T* X,
    T* Y) {
  const int X_HxW = X_D * X_H * X_W;
  const int Y_HxW = Y_D * Y_H * Y_W;
  const int n = blockIdx.x / Y_HxW;
  const int y = blockIdx.x % Y_HxW;
  const int yy = y / Y_W;
  const int yw = y % Y_W;
  const int yh = yy % Y_H;
  const int yd = yy / Y_H;
  const int xd = yd * stride_d;
  const int xh = yh * stride_h;
  const int xw = yw * stride_w;
  const int p = max(xd - pad_p, 0);
  const int a = min(xd - pad_p + kernel_d, X_D);
  const int t = max(xh - pad_t, 0);
  const int b = min(xh - pad_t + kernel_h, X_H);
  const int l = max(xw - pad_l, 0);
  const int r = min(xw - pad_l + kernel_w, X_W);
  const T scale = T(1) /
      static_cast<T>(count_include_pad ? kernel_d * kernel_h * kernel_w
                                       : (a - p) * (b - t) * (r - l));
  const T* X_ptr = X + n * X_HxW * C;
  T* Y_ptr = Y + n * Y_HxW * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T sum = 0;
    for (int i = p; i < a; ++i) {
      for (int j = t; j < b; ++j) {
        for (int k = l; k < r; ++k) {
          sum += X_ptr[((i * X_H + j) * X_W + k) * C + c];
        }
      }
    }
    Y_ptr[y * C + c] = sum * scale;
  }
}

template <typename T>
__global__ void Ave1DPoolBackwardNCHW(
    const int nthreads,
    const T* const top_diff,
    const int num,
    const int channels,
    const int height,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int h = index % height + pad_t;
    const int c = (index / height) % channels;
    const int n = index / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    T gradient = 0;
    const T* const top_diff_slice =
        top_diff + (n * channels + c) * pooled_height;
    for (int ph = phstart; ph < phend; ++ph) {
      // figure out the pooling size
      int hstart = ph * stride_h - pad_t;
      int hend = min(hstart + kernel_h, height);
      hstart = max(hstart, 0);
      int pool_size = (hend - hstart);
      gradient += top_diff_slice[ph] / pool_size;
    }
    bottom_diff[index] = gradient;
  }
}

template <typename T>
__global__ void Ave2DPoolBackwardNCHW(
    const int nthreads,
    const T* const top_diff,
    const int num,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width + pad_l;
    const int h = (index / width) % height + pad_t;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    T gradient = 0;
    const T* const top_diff_slice =
        top_diff + (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_t;
        int wstart = pw * stride_w - pad_l;
        int hend = min(hstart + kernel_h, height);
        int wend = min(wstart + kernel_w, width);
        hstart = max(hstart, 0);
        wstart = max(wstart, 0);
        int pool_size = (hend - hstart) * (wend - wstart);
        gradient += top_diff_slice[ph * pooled_width + pw] / pool_size;
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename T>
__global__ void Ave3DPoolBackwardNCHW(
    const int nthreads,
    const T* const top_diff,
    const int num,
    const int channels,
    const int height,
    const int width,
    const int depth,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int d = index % depth + pad_f;
    const int w = (index / depth) % width + pad_l;
    const int h = (index / depth / width) % height + pad_t;
    const int c = (index / depth / width / height) % channels;
    const int n = index / depth / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    const int pdstart = (d < kernel_d) ? 0 : (d - kernel_d) / stride_d + 1;
    const int pdend = min(d / stride_d + 1, pooled_depth);
    T gradient = 0;
    const T* const top_diff_slice = top_diff +
        (n * channels + c) * pooled_height * pooled_width * pooled_depth;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        for (int pd = pdstart; pd < pdend; ++pd) {
          // figure out the pooling size
          int hstart = ph * stride_h - pad_t;
          int wstart = pw * stride_w - pad_l;
          int dstart = pd * stride_d - pad_f;
          int hend = min(hstart + kernel_h, height);
          int wend = min(wstart + kernel_w, width);
          int dend = min(dstart + kernel_d, depth);
          hstart = max(hstart, 0);
          wstart = max(wstart, 0);
          dstart = max(dstart, 0);
          int pool_size = (hend - hstart) * (wend - wstart) * (dend - dstart);
          const int pooled_index =
              ph * pooled_depth * pooled_width + pooled_depth * pw + pd;
          gradient += top_diff_slice[pooled_index] / pool_size;
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename T>
__global__ void Ave1DPoolBackwardNHWC(
    const int nthreads,
    const T* const top_diff,
    const int num,
    const int height,
    const int channels,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int c = index % channels;
    const int h = (index / channels) % height + pad_t;
    const int n = index / channels / height;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    T gradient = 0;
    const T* const top_diff_slice = top_diff + n * pooled_height * channels + c;
    for (int ph = phstart; ph < phend; ++ph) {
      // figure out the pooling size
      int hstart = ph * stride_h - pad_t;
      int hend = min(hstart + kernel_h, height);
      hstart = max(hstart, 0);
      int pool_size = (hend - hstart);
      gradient += top_diff_slice[ph * channels] / pool_size;
    }
    bottom_diff[index] = gradient;
  }
}

template <typename T>
__global__ void Ave2DPoolBackwardNHWC(
    const int nthreads,
    const T* const top_diff,
    const int num,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int c = index % channels;
    const int w = index / channels % width + pad_l;
    const int h = (index / channels / width) % height + pad_t;
    const int n = index / channels / width / height;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    T gradient = 0;
    const T* const top_diff_slice =
        top_diff + n * pooled_height * pooled_width * channels + c;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_t;
        int wstart = pw * stride_w - pad_l;
        int hend = min(hstart + kernel_h, height);
        int wend = min(wstart + kernel_w, width);
        hstart = max(hstart, 0);
        wstart = max(wstart, 0);
        int pool_size = (hend - hstart) * (wend - wstart);
        gradient +=
            top_diff_slice[(ph * pooled_width + pw) * channels] / pool_size;
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename T>
__global__ void Ave3DPoolBackwardNHWC(
    const int nthreads,
    const T* const top_diff,
    const int num,
    const int height,
    const int width,
    const int depth,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int c = index % channels;
    const int d = index / channels % depth + pad_f;
    const int w = (index / channels / depth) % width + pad_l;
    const int h = (index / channels / depth / width) % height + pad_t;
    const int n = index / channels / depth / width / height;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    const int pdstart = (d < kernel_d) ? 0 : (d - kernel_d) / stride_d + 1;
    const int pdend = min(d / stride_d + 1, pooled_depth);
    T gradient = 0;
    const T* const top_diff_slice = top_diff +
        n * pooled_height * pooled_width * pooled_depth * channels + c;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        for (int pd = pdstart; pd < pdend; ++pd) {
          // figure out the pooling size
          int hstart = ph * stride_h - pad_t;
          int wstart = pw * stride_w - pad_l;
          int dstart = pd * stride_d - pad_f;
          int hend = min(hstart + kernel_h, height);
          int wend = min(wstart + kernel_w, width);
          int dend = min(dstart + kernel_d, depth);
          hstart = max(hstart, 0);
          wstart = max(wstart, 0);
          dstart = max(dstart, 0);
          int pool_size = (hend - hstart) * (wend - wstart) * (dend - dstart);
          const int pooled_index =
              (ph * pooled_depth * pooled_width + pw * pooled_depth + pd) *
              channels;
          gradient += top_diff_slice[pooled_index] / pool_size;
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}

} // namespace

template <>
template <>
bool AveragePoolFunctor<HIPContext>::Forward<float, StorageOrder::NCHW>(
    const int N,
    const int C,
    const std::vector<int>& X_dims,
    const std::vector<int>& Y_dims,
    const std::vector<int>& kernel,
    const std::vector<int>& /* dilation */,
    const std::vector<int>& stride,
    const std::vector<int>& pads,
    const float* X,
    float* Y,
    HIPContext* context) const {
  const int ndim = X_dims.size();
  const int Y_HxW = std::accumulate(
      Y_dims.cbegin(), Y_dims.cend(), 1, std::multiplies<int>());
  const int K = (Y_HxW + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS;
  switch (ndim) {
    case 1: {
      AveragePool1DForwardNCHWCUDAKernel<float>
          <<<N * C * K, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              K,
              X_dims[0],
              Y_dims[0],
              kernel[0],
              stride[0],
              pads[0],
              count_include_pad,
              X,
              Y);
      return true;
    }
    case 2: {
      AveragePool2DForwardNCHWCUDAKernel<float>
          <<<N * C * K, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              K,
              X_dims[0],
              X_dims[1],
              Y_dims[0],
              Y_dims[1],
              kernel[0],
              kernel[1],
              stride[0],
              stride[1],
              pads[0],
              pads[1],
              count_include_pad,
              X,
              Y);
      return true;
    }
    case 3: {
      AveragePool3DForwardNCHWCUDAKernel<float>
          <<<N * C * K, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              K,
              X_dims[0],
              X_dims[1],
              X_dims[2],
              Y_dims[0],
              Y_dims[1],
              Y_dims[2],
              kernel[0],
              kernel[1],
              kernel[2],
              stride[0],
              stride[1],
              stride[2],
              pads[0],
              pads[1],
              pads[2],
              count_include_pad,
              X,
              Y);
      return true;
    }
    default: {
      CAFFE_THROW("Unsupported pooling dim: ", ndim);
      return false;
    }
  }
}

template <>
template <>
bool AveragePoolFunctor<HIPContext>::Forward<float, StorageOrder::NHWC>(
    const int N,
    const int C,
    const std::vector<int>& X_dims,
    const std::vector<int>& Y_dims,
    const std::vector<int>& kernel,
    const std::vector<int>& /* dilation */,
    const std::vector<int>& stride,
    const std::vector<int>& pads,
    const float* X,
    float* Y,
    HIPContext* context) const {
  const int ndim = X_dims.size();
  const int Y_HxW = std::accumulate(
      Y_dims.cbegin(), Y_dims.cend(), 1, std::multiplies<int>());
  switch (ndim) {
    case 1: {
      AveragePool1DForwardNHWCCUDAKernel<float>
          <<<N * Y_HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              C,
              X_dims[0],
              Y_dims[0],
              kernel[0],
              stride[0],
              pads[0],
              count_include_pad,
              X,
              Y);
      return true;
    }
    case 2: {
      AveragePool2DForwardNHWCCUDAKernel<float>
          <<<N * Y_HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              C,
              X_dims[0],
              X_dims[1],
              Y_dims[0],
              Y_dims[1],
              kernel[0],
              kernel[1],
              stride[0],
              stride[1],
              pads[0],
              pads[1],
              count_include_pad,
              X,
              Y);
      return true;
    }
    case 3: {
      AveragePool3DForwardNHWCCUDAKernel<float>
          <<<N * Y_HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              C,
              X_dims[0],
              X_dims[1],
              X_dims[2],
              Y_dims[0],
              Y_dims[1],
              Y_dims[2],
              kernel[0],
              kernel[1],
              kernel[2],
              stride[0],
              stride[1],
              stride[2],
              pads[0],
              pads[1],
              pads[2],
              count_include_pad,
              X,
              Y);
      return true;
    }
    default: {
      CAFFE_THROW("Unsupported pooling dim: ", ndim);
      return false;
    }
  }
}

template <>
bool PoolGradientOp<float, HIPContext, AveragePool>::
    RunOnDeviceWithOrderNCHW() {
  auto& X = Input(0);
  auto& dY = Input(2);
  CAFFE_ENFORCE_EQ(dY.dim32(1), X.dim32(1));
  auto* dX = Output(0);
  dX->ResizeLike(X);
  vector<int> dims(X.sizes().begin() + 2, X.sizes().end());
  ConvPoolOpBase<HIPContext>::ComputePads(dims);
  switch (kernel_.size()) {
    case 1:
      Ave1DPoolBackwardNCHW<float>
          <<<CAFFE_GET_BLOCKS(X.size()),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              X.size(),
              dY.data<float>(),
              X.dim32(0),
              X.dim32(1),
              X.dim32(2),
              dY.dim32(2),
              kernel_h(),
              stride_h(),
              pad_t(),
              dX->template mutable_data<float>());
      break;
    case 2:
      Ave2DPoolBackwardNCHW<float>
          <<<CAFFE_GET_BLOCKS(X.size()),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              X.size(),
              dY.data<float>(),
              X.dim32(0),
              X.dim32(1),
              X.dim32(2),
              X.dim32(3),
              dY.dim32(2),
              dY.dim32(3),
              kernel_h(),
              kernel_w(),
              stride_h(),
              stride_w(),
              pad_t(),
              pad_l(),
              dX->template mutable_data<float>());
      break;
    case 3:
      Ave3DPoolBackwardNCHW<float>
          <<<CAFFE_GET_BLOCKS(X.size()),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              X.size(),
              dY.data<float>(),
              X.dim32(0),
              X.dim32(1),
              X.dim32(2),
              X.dim32(3),
              X.dim32(4),
              dY.dim32(2),
              dY.dim32(3),
              dY.dim32(4),
              kernel_h(),
              kernel_w(),
              kernel_[2],
              stride_h(),
              stride_w(),
              stride_[2],
              pad_t(),
              pad_l(),
              pads_[2],
              dX->template mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}

template <>
bool PoolGradientOp<float, HIPContext, AveragePool>::
    RunOnDeviceWithOrderNHWC() {
  auto& X = Input(0);
  auto& dY = Input(2);
  CAFFE_ENFORCE_EQ(X.ndim(), dY.ndim());
  CAFFE_ENFORCE_EQ(X.dim32(X.ndim() - 1), dY.dim32(dY.ndim() - 1));
  auto* dX = Output(0);
  dX->ResizeLike(X);
  vector<int> dims(X.sizes().begin() + 1, X.sizes().end() - 1);
  ConvPoolOpBase<HIPContext>::ComputePads(dims);
  switch (kernel_.size()) {
    case 1:
      Ave1DPoolBackwardNHWC<float>
          <<<CAFFE_GET_BLOCKS(X.size()),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              X.size(),
              dY.data<float>(),
              X.dim32(0),
              X.dim32(1),
              X.dim32(2),
              dY.dim32(1),
              kernel_h(),
              stride_h(),
              pad_t(),
              dX->template mutable_data<float>());
      break;
    case 2:
      Ave2DPoolBackwardNHWC<float>
          <<<CAFFE_GET_BLOCKS(X.size()),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              X.size(),
              dY.data<float>(),
              X.dim32(0),
              X.dim32(1),
              X.dim32(2),
              X.dim32(3),
              dY.dim32(1),
              dY.dim32(2),
              kernel_h(),
              kernel_w(),
              stride_h(),
              stride_w(),
              pad_t(),
              pad_l(),
              dX->template mutable_data<float>());
      break;
    case 3:
      Ave3DPoolBackwardNHWC<float>
          <<<CAFFE_GET_BLOCKS(X.size()),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              X.size(),
              dY.data<float>(),
              X.dim32(0),
              X.dim32(1),
              X.dim32(2),
              X.dim32(3),
              X.dim32(4),
              dY.dim32(1),
              dY.dim32(2),
              dY.dim32(3),
              kernel_h(),
              kernel_w(),
              kernel_[2],
              stride_h(),
              stride_w(),
              stride_[2],
              pad_t(),
              pad_l(),
              pads_[2],
              dX->template mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}

namespace {

template <typename T>
__global__ void MaxPool1DForwardNCHW(
    const int nthreads,
    const T* bottom_data,
    const int channels,
    const int height,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int ph = index % pooled_height;
    int c = (index / pooled_height) % channels;
    int n = index / pooled_height / channels;
    int hstart = ph * stride_h - pad_t;
    int hend = min(hstart + kernel_h, height);
    hstart = max(hstart, 0);
    T maxval = -FLT_MAX;
    const T* bdata_offset = bottom_data + n * channels * height;
    for (int h = hstart; h < hend; ++h) {
      int idx = c * height + h;
      if (bdata_offset[idx] > maxval) {
        maxval = bdata_offset[idx];
      }
    }
    top_data[index] = maxval;
  }
}

template <typename T>
__global__ void MaxPool2DForwardNCHW(
    const int nthreads,
    const T* bottom_data,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_t;
    int wstart = pw * stride_w - pad_l;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    T maxval = -FLT_MAX;
    const T* bdata_offset = bottom_data + n * channels * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int idx = c * height * width + h * width + w;
        if (bdata_offset[idx] > maxval) {
          maxval = bdata_offset[idx];
        }
      }
    }
    top_data[index] = maxval;
  }
}

template <typename T>
__global__ void MaxPool3DForwardNCHW(
    const int nthreads,
    const T* bottom_data,
    const int channels,
    const int height,
    const int width,
    const int depth,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int pd = index % pooled_depth;
    int pw = (index / pooled_depth) % pooled_width;
    int ph = (index / pooled_depth / pooled_width) % pooled_height;
    int c = (index / pooled_depth / pooled_width / pooled_height) % channels;
    int n = index / pooled_depth / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_t;
    int wstart = pw * stride_w - pad_l;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    int dstart = pd * stride_d - pad_f;
    int dend = min(dstart + kernel_d, depth);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    dstart = max(dstart, 0);
    T maxval = -FLT_MAX;
    const T* bdata_offset = bottom_data + n * channels * height * width * depth;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        for (int d = dstart; d < dend; ++d) {
          int idx = ((c * height + h) * width + w) * depth + d;
          if (bdata_offset[idx] > maxval) {
            maxval = bdata_offset[idx];
          }
        }
      }
    }
    top_data[index] = maxval;
  }
}

template <typename T>
__global__ void MaxPool1DForwardNHWC(
    const int nthreads,
    const T* bottom_data,
    const int height,
    const int channels,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index;
    int c = n % channels;
    n /= channels;
    int hstart = (n % pooled_height) * stride_h - pad_t;
    n /= pooled_height;
    int hend = min(hstart + kernel_h, height);
    hstart = max(hstart, 0);
    T maxval = -FLT_MAX;
    const T* bdata_offset = bottom_data + n * height * channels;
    for (int h = hstart; h < hend; ++h) {
      int idx = h * channels + c;
      if (bdata_offset[idx] > maxval) {
        maxval = bdata_offset[idx];
      }
    }
    top_data[index] = maxval;
  }
}

template <typename T>
__global__ void MaxPool2DForwardNHWC(
    const int nthreads,
    const T* bottom_data,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index;
    int c = n % channels;
    n /= channels;
    int wstart = (n % pooled_width) * stride_w - pad_l;
    n /= pooled_width;
    int hstart = (n % pooled_height) * stride_h - pad_t;
    n /= pooled_height;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    T maxval = -FLT_MAX;
    const T* bdata_offset = bottom_data + n * height * width * channels;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int idx = (h * width + w) * channels + c;
        if (bdata_offset[idx] > maxval) {
          maxval = bdata_offset[idx];
        }
      }
    }
    top_data[index] = maxval;
  }
}

template <typename T>
__global__ void MaxPool3DForwardNHWC(
    const int nthreads,
    const T* bottom_data,
    const int height,
    const int width,
    const int depth,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index;
    int c = n % channels;
    n /= channels;
    int dstart = (n % pooled_depth) * stride_d - pad_f;
    n /= pooled_depth;
    int wstart = (n % pooled_width) * stride_w - pad_l;
    n /= pooled_width;
    int hstart = (n % pooled_height) * stride_h - pad_t;
    n /= pooled_height;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    int dend = min(dstart + kernel_d, depth);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    dstart = max(dstart, 0);
    T maxval = -FLT_MAX;
    const T* bdata_offset = bottom_data + n * height * width * depth * channels;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        for (int d = dstart; d < dend; ++d) {
          int idx = ((h * width + w) * depth + d) * channels + c;
          if (bdata_offset[idx] > maxval) {
            maxval = bdata_offset[idx];
          }
        }
      }
    }
    top_data[index] = maxval;
  }
}

template <typename T>
__global__ void MaxPool1DBackwardNCHW(
    const int nthreads,
    const T* const bottom_data,
    const T* const top_data,
    const T* const top_diff,
    const int num,
    const int channels,
    const int height,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int h = index % height + pad_t;
    const int c = (index / height) % channels;
    const int n = index / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int top_offset = (n * channels + c) * pooled_height;
    bottom_diff[index] = 0;
    for (int ph = phstart; ph < phend; ++ph) {
      int top_local_offset = top_offset + ph;
      if (bottom_data[index] == top_data[top_local_offset]) {
        bottom_diff[index] += top_diff[top_local_offset];
      }
    }
  }
}

template <typename T>
__global__ void MaxPool2DBackwardNCHW(
    const int nthreads,
    const T* const bottom_data,
    const T* const top_data,
    const T* const top_diff,
    const int num,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width + pad_l;
    const int h = (index / width) % height + pad_t;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    const int top_offset = (n * channels + c) * pooled_height * pooled_width;
    bottom_diff[index] = 0;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        int top_local_offset = top_offset + ph * pooled_width + pw;
        if (bottom_data[index] == top_data[top_local_offset]) {
          bottom_diff[index] += top_diff[top_local_offset];
        }
      }
    }
  }
}

template <typename T>
__global__ void MaxPool3DBackwardNCHW(
    const int nthreads,
    const T* const bottom_data,
    const T* const top_data,
    const T* const top_diff,
    const int num,
    const int channels,
    const int height,
    const int width,
    const int depth,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int d = index % depth + pad_f;
    const int w = (index / depth) % width + pad_l;
    const int h = (index / depth / width) % height + pad_t;
    const int c = (index / depth / width / height) % channels;
    const int n = index / depth / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    const int pdstart = (d < kernel_d) ? 0 : (d - kernel_d) / stride_d + 1;
    const int pdend = min(d / stride_d + 1, pooled_depth);
    const int top_offset =
        (n * channels + c) * pooled_height * pooled_width * pooled_depth;
    bottom_diff[index] = 0;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        for (int pd = pdstart; pd < pdend; ++pd) {
          int top_local_offset =
              top_offset + (ph * pooled_width + pw) * pooled_depth + pd;
          if (bottom_data[index] == top_data[top_local_offset]) {
            bottom_diff[index] += top_diff[top_local_offset];
          }
        }
      }
    }
  }
}

template <typename T>
__global__ void MaxPool1DBackwardNHWC(
    const int nthreads,
    const T* const bottom_data,
    const T* const top_data,
    const T* const top_diff,
    const int num,
    const int height,
    const int channels,
    const int pooled_height,
    const int kernel_h,
    const int stride_h,
    const int pad_t,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int c = index % channels;
    const int h = (index / channels) % height + pad_t;
    const int n = index / channels / height;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int top_offset = n * pooled_height * channels + c;
    bottom_diff[index] = 0;
    for (int ph = phstart; ph < phend; ++ph) {
      int top_local_offset = top_offset + ph * channels;
      if (bottom_data[index] == top_data[top_local_offset]) {
        bottom_diff[index] += top_diff[top_local_offset];
      }
    }
  }
}

template <typename T>
__global__ void MaxPool2DBackwardNHWC(
    const int nthreads,
    const T* const bottom_data,
    const T* const top_data,
    const T* const top_diff,
    const int num,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int c = index % channels;
    const int w = index / channels % width + pad_l;
    const int h = (index / channels / width) % height + pad_t;
    const int n = index / channels / width / height;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    const int top_offset = n * pooled_height * pooled_width * channels + c;
    bottom_diff[index] = 0;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        int top_local_offset = top_offset + (ph * pooled_width + pw) * channels;
        if (bottom_data[index] == top_data[top_local_offset]) {
          bottom_diff[index] += top_diff[top_local_offset];
        }
      }
    }
  }
}

template <typename T>
__global__ void MaxPool3DBackwardNHWC(
    const int nthreads,
    const T* const bottom_data,
    const T* const top_data,
    const T* const top_diff,
    const int num,
    const int height,
    const int width,
    const int depth,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int pooled_depth,
    const int kernel_h,
    const int kernel_w,
    const int kernel_d,
    const int stride_h,
    const int stride_w,
    const int stride_d,
    const int pad_t,
    const int pad_l,
    const int pad_f,
    T* const bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int c = index % channels;
    const int d = index / channels % depth + pad_f;
    const int w = (index / depth / channels) % width + pad_l;
    const int h = (index / channels / depth / width) % height + pad_t;
    const int n = index / channels / depth / width / height;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    const int pdstart = (d < kernel_d) ? 0 : (d - kernel_d) / stride_d + 1;
    const int pdend = min(d / stride_d + 1, pooled_depth);
    const int top_offset =
        n * pooled_height * pooled_width * pooled_depth * channels + c;
    bottom_diff[index] = 0;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        for (int pd = pdstart; pd < pdend; ++pd) {
          int top_local_offset = top_offset +
              ((ph * pooled_width + pw) * pooled_depth + d) * channels;
          if (bottom_data[index] == top_data[top_local_offset]) {
            bottom_diff[index] += top_diff[top_local_offset];
          }
        }
      }
    }
  }
}
} // namespace

template <>
bool PoolOp<float, HIPContext, MaxPool>::RunOnDeviceWithOrderNCHW() {
  auto& X = Input(0);
  auto* Y = Output(0);
  ConvPoolOpBase<HIPContext>::SetOutputSize(X, Y, X.dim32(1));
  int output_size = Y->size();
  switch (kernel_.size()) {
    case 1:
      MaxPool1DForwardNCHW<float>
          <<<CAFFE_GET_BLOCKS(output_size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              output_size,
              X.data<float>(),
              X.dim32(1),
              X.dim32(2),
              Y->dim32(2),
              kernel_h(),
              stride_h(),
              pad_t(),
              Y->template mutable_data<float>());
      break;
    case 2:
      MaxPool2DForwardNCHW<float>
          <<<CAFFE_GET_BLOCKS(output_size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              output_size,
              X.data<float>(),
              X.dim32(1),
              X.dim32(2),
              X.dim32(3),
              Y->dim32(2),
              Y->dim32(3),
              kernel_h(),
              kernel_w(),
              stride_h(),
              stride_w(),
              pad_t(),
              pad_l(),
              Y->template mutable_data<float>());
      break;
    case 3:
      MaxPool3DForwardNCHW<float>
          <<<CAFFE_GET_BLOCKS(output_size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              output_size,
              X.data<float>(),
              X.dim32(1),
              X.dim32(2),
              X.dim32(3),
              X.dim32(4),
              Y->dim32(2),
              Y->dim32(3),
              Y->dim32(4),
              kernel_h(),
              kernel_w(),
              kernel_[2],
              stride_h(),
              stride_w(),
              stride_[2],
              pad_t(),
              pad_l(),
              pads_[2],
              Y->template mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}

template <>
bool PoolOp<float, HIPContext, MaxPool>::RunOnDeviceWithOrderNHWC() {
  auto& X = Input(0);
  auto* Y = Output(0);
  ConvPoolOpBase<HIPContext>::SetOutputSize(X, Y, X.dim32(X.ndim() - 1));
  int output_size = Y->size();
  switch (kernel_.size()) {
    case 1:
      MaxPool1DForwardNHWC<float>
          <<<CAFFE_GET_BLOCKS(output_size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              output_size,
              X.data<float>(),
              X.dim32(1),
              X.dim32(2),
              Y->dim32(1),
              kernel_h(),
              stride_h(),
              pad_t(),
              Y->template mutable_data<float>());
      break;
    case 2:
      MaxPool2DForwardNHWC<float>
          <<<CAFFE_GET_BLOCKS(output_size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              output_size,
              X.data<float>(),
              X.dim32(1),
              X.dim32(2),
              X.dim32(3),
              Y->dim32(1),
              Y->dim32(2),
              kernel_h(),
              kernel_w(),
              stride_h(),
              stride_w(),
              pad_t(),
              pad_l(),
              Y->template mutable_data<float>());
      break;
    case 3:
      MaxPool3DForwardNHWC<float>
          <<<CAFFE_GET_BLOCKS(output_size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              output_size,
              X.data<float>(),
              X.dim32(1),
              X.dim32(2),
              X.dim32(3),
              X.dim32(4),
              Y->dim32(1),
              Y->dim32(2),
              Y->dim32(3),
              kernel_h(),
              kernel_w(),
              kernel_[2],
              stride_h(),
              stride_w(),
              stride_[2],
              pad_t(),
              pad_l(),
              pads_[2],
              Y->template mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}

template <>
bool PoolGradientOp<float, HIPContext, MaxPool>::RunOnDeviceWithOrderNCHW() {
  auto& X = Input(0);
  auto& Y = Input(1);
  auto& dY = Input(2);
  CAFFE_ENFORCE_EQ(dY.ndim(), X.ndim());
  auto* dX = Output(0);
  dX->ResizeLike(X);
  vector<int> dims(X.sizes().begin() + 2, X.sizes().end());
  ConvPoolOpBase<HIPContext>::ComputePads(dims);
  switch (kernel_.size()) {
    case 1:
      MaxPool1DBackwardNCHW<float>
          <<<CAFFE_GET_BLOCKS(X.size()),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              X.size(),
              X.data<float>(),
              Y.data<float>(),
              dY.data<float>(),
              X.dim32(0),
              X.dim32(1),
              X.dim32(2),
              dY.dim32(2),
              kernel_h(),
              stride_h(),
              pad_t(),
              dX->template mutable_data<float>());
      break;
    case 2:
      MaxPool2DBackwardNCHW<float>
          <<<CAFFE_GET_BLOCKS(X.size()),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              X.size(),
              X.data<float>(),
              Y.data<float>(),
              dY.data<float>(),
              X.dim32(0),
              X.dim32(1),
              X.dim32(2),
              X.dim32(3),
              dY.dim32(2),
              dY.dim32(3),
              kernel_h(),
              kernel_w(),
              stride_h(),
              stride_w(),
              pad_t(),
              pad_l(),
              dX->template mutable_data<float>());
      break;
    case 3:
      MaxPool3DBackwardNCHW<float>
          <<<CAFFE_GET_BLOCKS(X.size()),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              X.size(),
              X.data<float>(),
              Y.data<float>(),
              dY.data<float>(),
              X.dim32(0),
              X.dim32(1),
              X.dim32(2),
              X.dim32(3),
              X.dim32(4),
              dY.dim32(2),
              dY.dim32(3),
              dY.dim32(4),
              kernel_h(),
              kernel_w(),
              kernel_[2],
              stride_h(),
              stride_w(),
              stride_[2],
              pad_t(),
              pad_l(),
              pads_[2],
              dX->template mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}

template <>
bool PoolGradientOp<float, HIPContext, MaxPool>::RunOnDeviceWithOrderNHWC() {
  auto& X = Input(0);
  auto& Y = Input(1);
  auto& dY = Input(2);
  CAFFE_ENFORCE_EQ(dY.ndim(), X.ndim());
  auto* dX = Output(0);
  dX->ResizeLike(X);
  vector<int> dims(X.sizes().begin() + 1, X.sizes().end() - 1);
  ConvPoolOpBase<HIPContext>::ComputePads(dims);
  switch (kernel_.size()) {
    case 1:
      MaxPool1DBackwardNHWC<float>
          <<<CAFFE_GET_BLOCKS(X.size()),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              X.size(),
              X.data<float>(),
              Y.data<float>(),
              dY.data<float>(),
              X.dim32(0),
              X.dim32(1),
              X.dim32(2),
              dY.dim32(1),
              kernel_h(),
              stride_h(),
              pad_t(),
              dX->template mutable_data<float>());
      break;
    case 2:
      MaxPool2DBackwardNHWC<float>
          <<<CAFFE_GET_BLOCKS(X.size()),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              X.size(),
              X.data<float>(),
              Y.data<float>(),
              dY.data<float>(),
              X.dim32(0),
              X.dim32(1),
              X.dim32(2),
              X.dim32(3),
              dY.dim32(1),
              dY.dim32(2),
              kernel_h(),
              kernel_w(),
              stride_h(),
              stride_w(),
              pad_t(),
              pad_l(),
              dX->template mutable_data<float>());
      break;
    case 3:
      MaxPool3DBackwardNHWC<float>
          <<<CAFFE_GET_BLOCKS(X.size()),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(
              X.size(),
              X.data<float>(),
              Y.data<float>(),
              dY.data<float>(),
              X.dim32(0),
              X.dim32(1),
              X.dim32(2),
              X.dim32(3),
              X.dim32(4),
              dY.dim32(1),
              dY.dim32(2),
              dY.dim32(3),
              kernel_h(),
              kernel_w(),
              kernel_[2],
              stride_h(),
              stride_w(),
              stride_[2],
              pad_t(),
              pad_l(),
              pads_[2],
              dX->template mutable_data<float>());
      break;
    default:
      CAFFE_THROW("Unsupported pooling size : ", kernel_.size());
  }
  return true;
}

REGISTER_HIP_OPERATOR(
    AveragePool,
    PoolOp<float, HIPContext, AveragePoolFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    AveragePoolGradient,
    PoolGradientOp<float, HIPContext, AveragePool>);

REGISTER_HIP_OPERATOR(
    AveragePool1D,
    PoolOp<float, HIPContext, AveragePoolFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    AveragePool1DGradient,
    PoolGradientOp<float, HIPContext, AveragePool>);

REGISTER_HIP_OPERATOR(
    AveragePool2D,
    PoolOp<float, HIPContext, AveragePoolFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    AveragePool2DGradient,
    PoolGradientOp<float, HIPContext, AveragePool>);

REGISTER_HIP_OPERATOR(
    AveragePool3D,
    PoolOp<float, HIPContext, AveragePoolFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    AveragePool3DGradient,
    PoolGradientOp<float, HIPContext, AveragePool>);

REGISTER_HIP_OPERATOR(MaxPool, PoolOp<float, HIPContext, MaxPool>);
REGISTER_HIP_OPERATOR(
    MaxPoolGradient,
    PoolGradientOp<float, HIPContext, MaxPool>);

REGISTER_HIP_OPERATOR(MaxPool1D, PoolOp<float, HIPContext, MaxPool>);
REGISTER_HIP_OPERATOR(
    MaxPool1DGradient,
    PoolGradientOp<float, HIPContext, MaxPool>);

REGISTER_HIP_OPERATOR(MaxPool2D, PoolOp<float, HIPContext, MaxPool>);
REGISTER_HIP_OPERATOR(
    MaxPool2DGradient,
    PoolGradientOp<float, HIPContext, MaxPool>);

REGISTER_HIP_OPERATOR(MaxPool3D, PoolOp<float, HIPContext, MaxPool>);
REGISTER_HIP_OPERATOR(
    MaxPool3DGradient,
    PoolGradientOp<float, HIPContext, MaxPool>);
} // namespace caffe2
